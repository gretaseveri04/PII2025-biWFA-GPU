#include "hip/hip_runtime.h"
extern "C" {
	#include "wavefront/wavefront_align.h"
}

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <limits.h>
#include "headers/commons.h"
#include "headers/biWFA.h"
#include <chrono>

#define CHECK(call)                                                                     \
{                                                                                     \
	const hipError_t err = call;                                                     \
	if (err != hipSuccess)                                                           \
	{                                                                                 \
		printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__); \
		exit(EXIT_FAILURE);                                                           \
	}                                                                                 \
}

#define CHECK_KERNELCALL()                                                            \
{                                                                                     \
	const hipError_t err = hipGetLastError();                                       \
	if (err != hipSuccess)                                                           \
	{                                                                                 \
		printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__); \
		exit(EXIT_FAILURE);                                                           \
	}                                                                                 \
}

__device__ void extend_max(bool *finish, const int score, int32_t *max_ak, wf_components_t *wf, const int max_score_scope, const int alignment_k, const int32_t alignment_offset, const int pattern_len) {
    if (wf->mwavefronts[score%num_wavefronts].offsets == NULL) {
        if (wf->alignment.num_null_steps > max_score_scope) {
            *finish = true;
        } else {
            *finish = false;
        }
    } else {
        // wavefront_extend_matches_packed_end2end_max()
        bool end_reached = false;
        int32_t max_antidiag_loc = 0;
        
        // Iterate over all wavefront offsets
        int k_start = wf->mwavefronts[score%num_wavefronts].lo;
        int k_end = wf->mwavefronts[score%num_wavefronts].hi;
        
        int tidx = threadIdx.x;
        
        for (int k = k_start; k <= k_end; ++k) {
            int32_t offset = wf->mwavefronts[score%num_wavefronts].offsets[k];
            if (offset == OFFSET_NULL) continue;
            
            // wavefront_extend_matches_kernel_blockwise() or wavefront_extend_matches_kernel()
            int equal_chars = 0;
            for (int i = offset; i < pattern_len; i++) {
                if((i - k) >= 0 && (i - k) < pattern_len) {
                    if (wf->alignment.pattern[i - k] == wf->alignment.text[i]) {
                        equal_chars++;
                    } else break;
                }
            }
            offset += equal_chars;
            
            // Return extended offset
            wf->mwavefronts[score%num_wavefronts].offsets[k] = offset;
            
            // Calculate antidiagonal and update max if needed
            int32_t antidiag = (2 * offset) - k;
            if (max_antidiag_loc < antidiag) {
                max_antidiag_loc = antidiag;
            }
        }
        
        // Update the max antidiagonal location
        *max_ak = max_antidiag_loc;
        
        // wavefront_termination_end2end()
        if (wf->mwavefronts[score%num_wavefronts].lo > alignment_k || alignment_k > wf->mwavefronts[score%num_wavefronts].hi) {
            end_reached = false;
        } else {
            int32_t moffset = wf->mwavefronts[score%num_wavefronts].offsets[alignment_k];
            if (moffset < alignment_offset) {
                end_reached = false;
            } else {
                end_reached = true;
            }
        }
        
        *finish = end_reached;
    }
}

__device__ void extend(bool *finish, const int score, const wf_components_t *wf, const int max_score_scope, const int alignment_k, const int32_t alignment_offset, const int pattern_len) {
    wf_t *mwf = &wf->mwavefronts[score % num_wavefronts];
    
    if (mwf->offsets == NULL) {
        *finish = (wf->alignment.num_null_steps > max_score_scope);
        return;
    }

    int lo = mwf->lo;
    int hi = mwf->hi;
    int k = lo + threadIdx.x;

    int32_t offset = 0;
    if (k <= hi) {
        offset = mwf->offsets[k];

        for (int i = offset; i < pattern_len; ++i) {
            int pattern_pos = i - k;
            int text_pos = i;

            if (pattern_pos < 0 || pattern_pos >= pattern_len) break;
            if (wf->alignment.pattern[pattern_pos] != wf->alignment.text[text_pos]) break;

            ++offset;
        }

        mwf->offsets[k] = offset;
    }

    __syncthreads();

    if (threadIdx.x == 0) {
        bool end_reached = false;
        if (alignment_k >= lo && alignment_k <= hi) {
            int32_t moffset = mwf->offsets[alignment_k];
            end_reached = (moffset >= alignment_offset);
        }
        *finish = end_reached;
    }
}

__device__ void nextWF(int *score, wf_components_t *wf, const bool forward, const int max_score_scope, const int text_len, const int pattern_len, int32_t *matrix_wf_m_g, int32_t *matrix_wf_i_g, int32_t *matrix_wf_d_g) {
    // Compute next (s+1) wavefront
    ++(*score);

    int score_mod = *score%num_wavefronts;

    // wavefront_compute_affine()
    int mismatch = *score - penalty_mismatch;
    int gap_open = *score - penalty_gap_open - penalty_gap_ext;
    int gap_extend = *score - penalty_gap_ext;

    // wavefront_compute_get_mwavefront()
    if((*score / num_wavefronts) > 0) {
        // Resetting old wavefronts' values
        wf->mwavefronts[score_mod].lo = -1;
        wf->mwavefronts[score_mod].hi = 1;
        wf->iwavefronts[score_mod].lo = -1;
        wf->iwavefronts[score_mod].hi = 1;
        wf->dwavefronts[score_mod].lo = -1;
        wf->dwavefronts[score_mod].hi = 1;
    }
    wf->mwavefronts[score_mod].offsets = matrix_wf_m_g + (num_wavefronts * wf_length * blockIdx.x) + (score_mod * wf_length) + wf_length/2;
    wf->mwavefronts[score_mod].null = false;
    wf->iwavefronts[score_mod].offsets = matrix_wf_i_g + (num_wavefronts * wf_length * blockIdx.x) + (score_mod * wf_length) + wf_length/2;
    wf->iwavefronts[score_mod].null = false;
    wf->dwavefronts[score_mod].offsets = matrix_wf_d_g + (num_wavefronts * wf_length * blockIdx.x) + (score_mod * wf_length) + wf_length/2;
    wf->dwavefronts[score_mod].null = false;

    wf_t in_mwavefront_misms = (mismatch < 0 || wf->mwavefronts[mismatch%num_wavefronts].offsets == NULL || wf->mwavefronts[mismatch%num_wavefronts].null) ? wf->wavefront_null : wf->mwavefronts[mismatch%num_wavefronts];
    wf_t in_mwavefront_open = (gap_open < 0 || wf->mwavefronts[gap_open%num_wavefronts].offsets == NULL || wf->mwavefronts[gap_open%num_wavefronts].null) ? wf->wavefront_null : wf->mwavefronts[gap_open%num_wavefronts];
    wf_t in_iwavefront_ext = (gap_extend < 0 || wf->iwavefronts[gap_extend%num_wavefronts].offsets == NULL || wf->iwavefronts[gap_extend%num_wavefronts].null) ? wf->wavefront_null : wf->iwavefronts[gap_extend%num_wavefronts];
    wf_t in_dwavefront_ext = (gap_extend < 0 || wf->dwavefronts[gap_extend%num_wavefronts].offsets == NULL || wf->dwavefronts[gap_extend%num_wavefronts].null) ? wf->wavefront_null : wf->dwavefronts[gap_extend%num_wavefronts];

    if (in_mwavefront_misms.null && in_mwavefront_open.null && in_iwavefront_ext.null && in_dwavefront_ext.null) {
        // wavefront_compute_allocate_output_null()
        wf->alignment.num_null_steps++; // Increment null-steps
        // Nullify Wavefronts
        wf->mwavefronts[score_mod].null = true;
        wf->iwavefronts[score_mod].null = true;
        wf->dwavefronts[score_mod].null = true;
    } else {
        wf->alignment.num_null_steps = 0;
        int hi, lo;

        // wavefront_compute_limits_input()
        int min_lo = in_mwavefront_misms.lo;
        int max_hi = in_mwavefront_misms.hi;

        if (!in_mwavefront_open.null && min_lo > (in_mwavefront_open.lo - 1)) min_lo = in_mwavefront_open.lo - 1;
        if (!in_mwavefront_open.null && max_hi < (in_mwavefront_open.hi + 1)) max_hi = in_mwavefront_open.hi + 1;
        if (!in_iwavefront_ext.null && min_lo > (in_iwavefront_ext.lo + 1)) min_lo = in_iwavefront_ext.lo + 1;
        if (!in_iwavefront_ext.null && max_hi < (in_iwavefront_ext.hi + 1)) max_hi = in_iwavefront_ext.hi + 1;
        if (!in_dwavefront_ext.null && min_lo > (in_dwavefront_ext.lo - 1)) min_lo = in_dwavefront_ext.lo - 1;
        if (!in_dwavefront_ext.null && max_hi < (in_dwavefront_ext.hi - 1)) max_hi = in_dwavefront_ext.hi - 1;
        lo = min_lo;
        hi = max_hi;

        // wavefront_compute_allocate_output()
        int effective_lo = lo;
        int effective_hi = hi;

        // wavefront_compute_limits_output()
        int eff_lo = effective_lo - (max_score_scope + 1);
        int eff_hi = effective_hi + (max_score_scope + 1);
        effective_lo = MIN(eff_lo, wf->alignment.historic_min_lo);
        effective_hi = MAX(eff_hi, wf->alignment.historic_max_hi);
        wf->alignment.historic_min_lo = effective_lo;
        wf->alignment.historic_max_hi = effective_hi;

        // Allocate M-Wavefront
        wf->mwavefronts[score_mod].lo = lo;
        wf->mwavefronts[score_mod].hi = hi;
        // Allocate I1-Wavefront
        if (!in_mwavefront_open.null || !in_iwavefront_ext.null) {
            wf->iwavefronts[score_mod].lo = lo;
            wf->iwavefronts[score_mod].hi = hi;
        } else {
            wf->iwavefronts[score_mod].null = true;
        }
        // Allocate D1-Wavefront
        if (!in_mwavefront_open.null || !in_dwavefront_ext.null) {
            wf->dwavefronts[score_mod].lo = lo;
            wf->dwavefronts[score_mod].hi = hi;
        } else {
            wf->dwavefronts[score_mod].null = true;
        }

        // wavefront_compute_init_ends()
        // Init wavefront ends
        bool m_misms_null = in_mwavefront_misms.null;
        bool m_gap_null = in_mwavefront_open.null;
        bool i_ext_null = in_iwavefront_ext.null;
        bool d_ext_null = in_dwavefront_ext.null;

        if (!m_misms_null) {
            // wavefront_compute_init_ends_wf_higher()
            if (in_mwavefront_misms.wf_elements_init_max >= hi) {
            } else {
                // Initialize lower elements
                int max_init = MAX(in_mwavefront_misms.wf_elements_init_max, in_mwavefront_misms.hi);
                int k;
                for (k = max_init + 1; k <= hi; ++k) {
                    in_mwavefront_misms.offsets[k] = OFFSET_NULL;
                }
                // Set new maximum
                in_mwavefront_misms.wf_elements_init_max = hi;
            }
            // wavefront_compute_init_ends_wf_lower()
            if (in_mwavefront_misms.wf_elements_init_min <= lo) {
            } else {
                // Initialize lower elements
                int min_init = MIN(in_mwavefront_misms.wf_elements_init_min, in_mwavefront_misms.lo);
                int k;
                for (k = lo; k < min_init; ++k) {
                    in_mwavefront_misms.offsets[k] = OFFSET_NULL;
                }
                // Set new minimum
                in_mwavefront_misms.wf_elements_init_min = lo;
            }
        }
        if (!m_gap_null) {
            // wavefront_compute_init_ends_wf_higher()
            if (in_mwavefront_open.wf_elements_init_max >= hi + 1) {
            } else {
                // Initialize lower elements
                int max_init = MAX(in_mwavefront_open.wf_elements_init_max, in_mwavefront_open.hi);
                int k;
                for (k = max_init + 1; k <= hi + 1; ++k) {
                    in_mwavefront_open.offsets[k] = OFFSET_NULL;
                }
                // Set new maximum
                in_mwavefront_open.wf_elements_init_max = hi + 1;
            }
            // wavefront_compute_init_ends_wf_lower()
            if (in_mwavefront_open.wf_elements_init_min <= lo - 1) {
            } else {
                // Initialize lower elements
                int min_init = MIN(in_mwavefront_open.wf_elements_init_min, in_mwavefront_open.lo);
                int k;
                for (k = lo - 1; k < min_init; ++k) {
                    in_mwavefront_open.offsets[k] = OFFSET_NULL;
                }
                // Set new minimum
                in_mwavefront_open.wf_elements_init_min = lo - 1;
            }
        }
        if (!i_ext_null) {
            // wavefront_compute_init_ends_wf_higher()
            if (in_iwavefront_ext.wf_elements_init_max >= hi) {
            } else {
                // Initialize lower elements
                int max_init = MAX(in_iwavefront_ext.wf_elements_init_max, in_iwavefront_ext.hi);
                int k;
                for (k = max_init + 1; k <= hi; ++k) {
                    in_iwavefront_ext.offsets[k] = OFFSET_NULL;
                }
                // Set new maximum
                in_iwavefront_ext.wf_elements_init_max = hi;
            }
            // wavefront_compute_init_ends_wf_lower()
            if (in_iwavefront_ext.wf_elements_init_min <= lo - 1) {
            } else {
                // Initialize lower elements
                int min_init = MIN(in_iwavefront_ext.wf_elements_init_min, in_iwavefront_ext.lo);
                int k;
                for (k = lo - 1; k < min_init; ++k) {
                    in_iwavefront_ext.offsets[k] = OFFSET_NULL;
                }
                // Set new minimum
                in_iwavefront_ext.wf_elements_init_min = lo - 1;
            }
        }
        if (!d_ext_null) {
            // wavefront_compute_init_ends_wf_higher()
            if (in_dwavefront_ext.wf_elements_init_max >= hi + 1) {
            } else {
                // Initialize lower elements
                int max_init = MAX(in_dwavefront_ext.wf_elements_init_max, in_dwavefront_ext.hi);
                int k;
                for (k = max_init + 1; k <= hi + 1; ++k) {
                    in_dwavefront_ext.offsets[k] = OFFSET_NULL;
                }
                // Set new maximum
                in_dwavefront_ext.wf_elements_init_max = hi + 1;
            }
            // wavefront_compute_init_ends_wf_lower()
            if (in_dwavefront_ext.wf_elements_init_min <= lo) {
            } else {
                // Initialize lower elements
                int min_init = MIN(in_dwavefront_ext.wf_elements_init_min, in_dwavefront_ext.lo);
                int k;
                for (k = lo; k < min_init; ++k) {
                    in_dwavefront_ext.offsets[k] = OFFSET_NULL;
                }
                // Set new minimum
                in_dwavefront_ext.wf_elements_init_min = lo;
            }
        }

        //wavefront_compute_affine_idm()
        // Compute-Next kernel loop
        int tidx = threadIdx.x;
        for (int i = lo; i <= hi; i += blockDim.x) {
            int idx = tidx + i;
            if (idx <= hi) {
                // Update I1
                int32_t ins_o = in_mwavefront_open.offsets[idx - 1];
                int32_t ins_e = in_iwavefront_ext.offsets[idx - 1];
                int32_t ins = MAX(ins_o, ins_e) + 1;
                wf->iwavefronts[score_mod].offsets[idx] = ins;

                // Update D1
                int32_t del_o = in_mwavefront_open.offsets[idx + 1];
                int32_t del_e = in_dwavefront_ext.offsets[idx + 1];
                int32_t del = MAX(del_o, del_e);
                wf->dwavefronts[score_mod].offsets[idx] = del;

                // Update M
                int32_t misms = in_mwavefront_misms.offsets[idx] + 1;
                int32_t max = MAX(del, MAX(misms, ins));

                // Adjust offset out of boundaries
                uint32_t h = max;
                uint32_t v = max - idx;
                if (h > text_len) max = OFFSET_NULL;
                if (v > pattern_len) max = OFFSET_NULL;
                wf->mwavefronts[score_mod].offsets[idx] = max;
            }
        }

        // wavefront_compute_process_ends()
        if (wf->mwavefronts[score_mod].offsets) {
            // wavefront_compute_trim_ends()
            int k;
            int lo = wf->mwavefronts[score_mod].lo;
            for (k = wf->mwavefronts[score_mod].hi; k >= lo; --k) {
                // Fetch offset
                int32_t offset = wf->mwavefronts[score_mod].offsets[k];
                // Check boundaries
                uint32_t h = offset; // Make unsigned to avoid checking negative
                uint32_t v = offset - k; // Make unsigned to avoid checking negative
                if (h <= text_len && v <= pattern_len) break;
            }
            wf->mwavefronts[score_mod].hi = k; // Set new hi
            wf->mwavefronts[score_mod].wf_elements_init_max = k;
            // Trim from lo
            int hi = wf->mwavefronts[score_mod].hi;
            for (k = wf->mwavefronts[score_mod].lo ; k <= hi; ++k) {
                // Fetch offset
                int32_t offset = wf->mwavefronts[score_mod].offsets[k];
                // Check boundaries
                uint32_t h = offset; // Make unsigned to avoid checking negative
                uint32_t v = offset - k; // Make unsigned to avoid checking negative
                if (h <= text_len && v <= pattern_len) break;
            }
            wf->mwavefronts[score_mod].lo = k; // Set new lo
            wf->mwavefronts[score_mod].wf_elements_init_min = k;
            wf->mwavefronts[score_mod].null = (wf->mwavefronts[score_mod].lo > wf->mwavefronts[score_mod].hi);
        }
        if (wf->iwavefronts[score_mod].offsets) {
            // wavefront_compute_trim_ends()
            int k;
            int lo = wf->iwavefronts[score_mod].lo;
            for (k = wf->iwavefronts[score_mod].hi; k >= lo; --k) {
                // Fetch offset
                int32_t offset = wf->iwavefronts[score_mod].offsets[k];
                // Check boundaries
                uint32_t h = offset; // Make unsigned to avoid checking negative
                uint32_t v = offset - k; // Make unsigned to avoid checking negative
                if (h <= text_len && v <= pattern_len) break;
            }
            wf->iwavefronts[score_mod].hi = k; // Set new hi
            wf->iwavefronts[score_mod].wf_elements_init_max = k;
            // Trim from lo
            int hi = wf->iwavefronts[score_mod].hi;
            for (k = wf->iwavefronts[score_mod].lo; k <= hi; ++k) {
                // Fetch offset
                int32_t offset = wf->iwavefronts[score_mod].offsets[k];
                // Check boundaries
                uint32_t h = offset; // Make unsigned to avoid checking negative
                uint32_t v = offset - k; // Make unsigned to avoid checking negative
                if (h <= text_len && v <= pattern_len) break;
            }
            wf->iwavefronts[score_mod].lo = k; // Set new lo
            wf->iwavefronts[score_mod].wf_elements_init_min = k;
            wf->iwavefronts[score_mod].null = (wf->iwavefronts[score_mod].lo > wf->iwavefronts[score_mod].hi);
        }
        if (wf->dwavefronts[score_mod].offsets) {
            // wavefront_compute_trim_ends()
            int k;
            int lo = wf->dwavefronts[score_mod].lo;
            for (k = wf->dwavefronts[score_mod].hi; k >= lo ; --k) {
                // Fetch offset
                int32_t offset = wf->dwavefronts[score_mod].offsets[k];
                // Check boundaries
                uint32_t h = offset; // Make unsigned to avoid checking negative
                uint32_t v = offset - k; // Make unsigned to avoid checking negative
                if (h <= text_len && v <= pattern_len) break;
            }
            wf->dwavefronts[score_mod].hi = k; // Set new hi
            wf->dwavefronts[score_mod].wf_elements_init_max = k;
            // Trim from lo
            int hi = wf->dwavefronts[score_mod].hi;
            for (k = wf->dwavefronts[score_mod].lo; k <= hi; ++k) {
                // Fetch offset
                int32_t offset = wf->dwavefronts[score_mod].offsets[k];
                // Check boundaries
                uint32_t h = offset; // Make unsigned to avoid checking negative
                uint32_t v = offset - k; // Make unsigned to avoid checking negative
                if (h <= text_len && v <= pattern_len) break;
            }
            wf->dwavefronts[score_mod].lo = k; // Set new lo
            wf->dwavefronts[score_mod].wf_elements_init_min = k;
            wf->dwavefronts[score_mod].null = (wf->dwavefronts[score_mod].lo > wf->dwavefronts[score_mod].hi);
        }
    }
}

__device__ void breakpoint_indel2indel(const int score_0, const int score_1, const wf_t *dwf_0, const wf_t *dwf_1, int *breakpoint_score, const int text_len, const int pattern_len) {
    int lo_0 = dwf_0->lo;
    int hi_0 = dwf_0->hi;
    int lo_1 = text_len - pattern_len - dwf_1->hi;
    int hi_1 = text_len - pattern_len - dwf_1->lo;

    if (hi_1 < lo_0 || hi_0 < lo_1) return;

    int min_hi = min(hi_0, hi_1);
    int max_lo = max(lo_0, lo_1);

    __shared__ int local_min[NUM_THREADS];
    int tid = threadIdx.x;
    local_min[tid] = INT_MAX;

    for (int k_0 = max_lo + tid; k_0 <= min_hi; k_0 += NUM_THREADS) {
        int k_1 = text_len - pattern_len - k_0;
        int dh_0 = dwf_0->offsets[k_0];
        int dh_1 = dwf_1->offsets[k_1];

        if ((dh_0 + dh_1) >= text_len) {
            int candidate = score_0 + score_1 - penalty_gap_open;
            if (candidate < local_min[tid]) {
                local_min[tid] = candidate;
            }
        }
    }

    __syncthreads();

    if (tid == 0) {
        int min_val = INT_MAX;
        for (int i = 0; i < NUM_THREADS; i++) {
            if (local_min[i] < min_val) {
                min_val = local_min[i];
            }
        }

        if (min_val < *breakpoint_score) {
            *breakpoint_score = min_val;
        }
    }
}

__device__ void breakpoint_m2m(const int score_0, const int score_1, const wf_t *mwf_0, const wf_t *mwf_1, int *breakpoint_score, const int text_len, const int pattern_len) {
    // Check wavefronts overlapping
    int lo_0 = mwf_0->lo;
    int hi_0 = mwf_0->hi;
    int lo_1 = text_len - pattern_len - mwf_1->hi;
    int hi_1 = text_len - pattern_len - mwf_1->lo;

    if (hi_1 < lo_0 || hi_0 < lo_1) return;
    
    // Compute overlapping interval
    int min_hi = MIN(hi_0, hi_1);
    int max_lo = MAX(lo_0, lo_1);
    int k_0;
    for (k_0 = max_lo; k_0 <= min_hi; k_0++) {
        const int k_1 = text_len - pattern_len - k_0;
        // Fetch offsets
        const int mh_0 = mwf_0->offsets[k_0];
        const int mh_1 = mwf_1->offsets[k_1];
        // Check breakpoint m2m
        if (mh_0 + mh_1 >= text_len && score_0 + score_1 < *breakpoint_score) {
            *breakpoint_score = score_0 + score_1; 
            return;
        }
    }
}

__device__ void overlap(const int score_0, const wf_components_t *wf_0, const int score_1, const wf_components_t *wf_1, const int max_score_scope, int *breakpoint_score, const int text_len, const int pattern_len) {
    // Fetch wavefront-0
    int score_mod_0 = score_0%num_wavefronts;
    wf_t *mwf_0 = &wf_0->mwavefronts[score_mod_0];

    if (mwf_0 == NULL) return;
    wf_t *d1wf_0 = &wf_0->dwavefronts[score_mod_0];
    wf_t *i1wf_0 = &wf_0->iwavefronts[score_mod_0];

    // Traverse all scores-1
    int i;
    for (i = 0; i < max_score_scope; ++i) {
        // Compute score
        const int score_i = score_1 - i;
        if (score_i < 0) break;
        int score_mod_i = score_i%num_wavefronts;

        if (score_0 + score_i - penalty_gap_open >= *breakpoint_score) continue;
        // Check breakpoint d2d
        wf_t *d1wf_1 = &wf_1->dwavefronts[score_mod_i];
        if (d1wf_0 != NULL && d1wf_1 != NULL) {
            breakpoint_indel2indel(score_0, score_i, d1wf_0, d1wf_1, breakpoint_score, text_len, pattern_len);
        }
        // Check breakpoint i2i
        wf_t *i1wf_1 = &wf_1->iwavefronts[score_mod_i];
        if (i1wf_0 != NULL && i1wf_1 != NULL) {
            breakpoint_indel2indel(score_0, score_i, i1wf_0, i1wf_1, breakpoint_score, text_len, pattern_len);
        }
        // Check M-breakpoints (indel, edit, gap-linear)
        if (score_0 + score_i >= *breakpoint_score) continue;
        wf_t *mwf_1 = &wf_1->mwavefronts[score_mod_i];
        if (mwf_1 != NULL) {
            breakpoint_m2m(score_0, score_i, mwf_0, mwf_1, breakpoint_score, text_len, pattern_len);
        }
    }
}

__global__ void biWFA_kernel(char *pattern_f_g, char *text_f_g, char *pattern_r_g, char *text_r_g, int *breakpoint_score_g, 
                      wf_t *mwavefronts_f, wf_t *iwavefronts_f, wf_t *dwavefronts_f, 
                      wf_t *mwavefronts_r, wf_t *iwavefronts_r, wf_t *dwavefronts_r,
                      const int lo_g, const int hi_g, int32_t *offsets_g, 
                      const int *pattern_len_array, const int *text_len_array, 
                      const int *pattern_offsets, const int *text_offsets,
                      const int max_score_scope, int32_t *matrix_wf_m_f,
                      int32_t *matrix_wf_i_f, int32_t *matrix_wf_d_f, 
                      int32_t *matrix_wf_m_r, int32_t *matrix_wf_i_r, int32_t *matrix_wf_d_r) {

    int lo = lo_g;
    int hi = hi_g;
    
    int pattern_len = pattern_len_array[blockIdx.x];
    int text_len = text_len_array[blockIdx.x];
    
    int pattern_offset = pattern_offsets[blockIdx.x];
    int text_offset = text_offsets[blockIdx.x];

    if (pattern_len + text_len > wf_length / 2 - 10) {
        if (threadIdx.x == 0) {
            *(breakpoint_score_g + blockIdx.x) = INT_MAX; 
        }
        return; 
    }

    int wf_matrix_size = num_wavefronts * wf_length;
    for (int i = 0; i < wf_matrix_size; i += blockDim.x) {
        int idx = i + threadIdx.x;
        if (idx < wf_matrix_size) {
            int block_offset = wf_matrix_size * blockIdx.x;
            *(matrix_wf_m_f + block_offset + idx) = OFFSET_NULL;
            *(matrix_wf_i_f + block_offset + idx) = OFFSET_NULL;
            *(matrix_wf_d_f + block_offset + idx) = OFFSET_NULL;
            *(matrix_wf_m_r + block_offset + idx) = OFFSET_NULL;
            *(matrix_wf_i_r + block_offset + idx) = OFFSET_NULL;
            *(matrix_wf_d_r + block_offset + idx) = OFFSET_NULL;
        }
    }
    
    __syncthreads();

    for (int i = 0; i < num_wavefronts; i += blockDim.x) {
        int idx = i + threadIdx.x;
        if (idx < num_wavefronts) {
            int block_offset = num_wavefronts * blockIdx.x;
            
            (mwavefronts_f + block_offset + idx)->null = true;
            (mwavefronts_f + block_offset + idx)->lo = 0;
            (mwavefronts_f + block_offset + idx)->hi = 0;
            (mwavefronts_f + block_offset + idx)->offsets = NULL;
            (mwavefronts_f + block_offset + idx)->wf_elements_init_max = 0;
            (mwavefronts_f + block_offset + idx)->wf_elements_init_min = 0;
            
            (mwavefronts_r + block_offset + idx)->null = true;
            (mwavefronts_r + block_offset + idx)->lo = 0;
            (mwavefronts_r + block_offset + idx)->hi = 0;
            (mwavefronts_r + block_offset + idx)->offsets = NULL;
            (mwavefronts_r + block_offset + idx)->wf_elements_init_max = 0;
            (mwavefronts_r + block_offset + idx)->wf_elements_init_min = 0;

            (iwavefronts_f + block_offset + idx)->null = true;
            (iwavefronts_f + block_offset + idx)->lo = 0;
            (iwavefronts_f + block_offset + idx)->hi = 0;
            (iwavefronts_f + block_offset + idx)->offsets = NULL;
            (iwavefronts_f + block_offset + idx)->wf_elements_init_max = 0;
            (iwavefronts_f + block_offset + idx)->wf_elements_init_min = 0;
            
            (iwavefronts_r + block_offset + idx)->null = true;
            (iwavefronts_r + block_offset + idx)->lo = 0;
            (iwavefronts_r + block_offset + idx)->hi = 0;
            (iwavefronts_r + block_offset + idx)->offsets = NULL;
            (iwavefronts_r + block_offset + idx)->wf_elements_init_max = 0;
            (iwavefronts_r + block_offset + idx)->wf_elements_init_min = 0;

            (dwavefronts_f + block_offset + idx)->null = true;
            (dwavefronts_f + block_offset + idx)->lo = 0;
            (dwavefronts_f + block_offset + idx)->hi = 0;
            (dwavefronts_f + block_offset + idx)->offsets = NULL;
            (dwavefronts_f + block_offset + idx)->wf_elements_init_max = 0;
            (dwavefronts_f + block_offset + idx)->wf_elements_init_min = 0;
            
            (dwavefronts_r + block_offset + idx)->null = true;
            (dwavefronts_r + block_offset + idx)->lo = 0;
            (dwavefronts_r + block_offset + idx)->hi = 0;
            (dwavefronts_r + block_offset + idx)->offsets = NULL;
            (dwavefronts_r + block_offset + idx)->wf_elements_init_max = 0;
            (dwavefronts_r + block_offset + idx)->wf_elements_init_min = 0;
        }
    }

    __syncthreads();

    if (threadIdx.x == 0) {
        wf_components_t wf_f, wf_r;
        wf_alignment_t alignment_f, alignment_r;
        int max_antidiag, score_f, score_r, forward_max_ak, reverse_max_ak, breakpoint_score, alignment_k;
        bool finish;

        alignment_f.pattern = pattern_f_g + pattern_offset;
        alignment_f.text = text_f_g + text_offset;
        alignment_f.historic_max_hi = 0;
        alignment_f.historic_min_lo = 0;
        wf_f.alignment = alignment_f;

        alignment_r.pattern = pattern_r_g + pattern_offset;
        alignment_r.text = text_r_g + text_offset;
        alignment_r.historic_max_hi = 0;
        alignment_r.historic_min_lo = 0;
        wf_r.alignment = alignment_r;

        wf_f.alignment.num_null_steps = 0;
        wf_f.alignment.historic_max_hi = hi;
        wf_f.alignment.historic_min_lo = lo;
        wf_r.alignment.num_null_steps = 0;
        wf_r.alignment.historic_max_hi = hi;
        wf_r.alignment.historic_min_lo = lo;

        int block_offset = num_wavefronts * blockIdx.x;
        wf_f.mwavefronts = mwavefronts_f + block_offset;
        wf_f.iwavefronts = iwavefronts_f + block_offset;
        wf_f.dwavefronts = dwavefronts_f + block_offset;

        int matrix_block_offset = num_wavefronts * wf_length * blockIdx.x;
        wf_f.mwavefronts[0].offsets = matrix_wf_m_f + matrix_block_offset + wf_length/2;
        wf_f.iwavefronts[0].offsets = matrix_wf_i_f + matrix_block_offset + wf_length/2;
        wf_f.dwavefronts[0].offsets = matrix_wf_d_f + matrix_block_offset + wf_length/2;

        wf_f.mwavefronts[0].null = false;
        wf_f.mwavefronts[0].lo = -1;
        wf_f.mwavefronts[0].hi = 1;
        wf_f.mwavefronts[0].offsets[-1] = OFFSET_NULL;  
        wf_f.mwavefronts[0].offsets[0] = 0;             
        wf_f.mwavefronts[0].offsets[1] = OFFSET_NULL;   
        wf_f.mwavefronts[0].wf_elements_init_min = 0;
        wf_f.mwavefronts[0].wf_elements_init_max = 0;

        wf_f.iwavefronts[0].null = true;
        wf_f.iwavefronts[0].lo = -1;
        wf_f.iwavefronts[0].hi = 1;
        wf_f.iwavefronts[0].wf_elements_init_min = 0;
        wf_f.iwavefronts[0].wf_elements_init_max = 0;

        wf_f.dwavefronts[0].null = true;
        wf_f.dwavefronts[0].lo = -1;
        wf_f.dwavefronts[0].hi = 1;
        wf_f.dwavefronts[0].wf_elements_init_min = 0;
        wf_f.dwavefronts[0].wf_elements_init_max = 0;

        wf_f.wavefront_null.null = true;
        wf_f.wavefront_null.lo = 1;
        wf_f.wavefront_null.hi = -1;
        wf_f.wavefront_null.offsets = offsets_g + wf_length/2;
        wf_f.wavefront_null.wf_elements_init_min = 0;
        wf_f.wavefront_null.wf_elements_init_max = 0;

        wf_r.mwavefronts = mwavefronts_r + block_offset;
        wf_r.iwavefronts = iwavefronts_r + block_offset;
        wf_r.dwavefronts = dwavefronts_r + block_offset;
        
        wf_r.mwavefronts[0].offsets = matrix_wf_m_r + matrix_block_offset + wf_length/2;
        wf_r.iwavefronts[0].offsets = matrix_wf_i_r + matrix_block_offset + wf_length/2;
        wf_r.dwavefronts[0].offsets = matrix_wf_d_r + matrix_block_offset + wf_length/2;

        wf_r.mwavefronts[0].null = false;
        wf_r.mwavefronts[0].lo = -1;
        wf_r.mwavefronts[0].hi = 1;
        wf_r.mwavefronts[0].offsets[-1] = OFFSET_NULL;  
        wf_r.mwavefronts[0].offsets[0] = 0;             
        wf_r.mwavefronts[0].offsets[1] = OFFSET_NULL;   
        wf_r.mwavefronts[0].wf_elements_init_min = 0;
        wf_r.mwavefronts[0].wf_elements_init_max = 0;

        wf_r.iwavefronts[0].null = true;
        wf_r.iwavefronts[0].lo = -1;
        wf_r.iwavefronts[0].hi = 1;
        wf_r.iwavefronts[0].wf_elements_init_min = 0;
        wf_r.iwavefronts[0].wf_elements_init_max = 0;

        wf_r.dwavefronts[0].null = true;
        wf_r.dwavefronts[0].lo = -1;
        wf_r.dwavefronts[0].hi = 1;
        wf_r.dwavefronts[0].wf_elements_init_min = 0;
        wf_r.dwavefronts[0].wf_elements_init_max = 0;

        wf_r.wavefront_null.null = true;
        wf_r.wavefront_null.lo = 1;
        wf_r.wavefront_null.hi = -1;
        wf_r.wavefront_null.offsets = offsets_g + wf_length/2;
        wf_r.wavefront_null.wf_elements_init_min = 0;
        wf_r.wavefront_null.wf_elements_init_max = 0;

        max_antidiag = text_len + pattern_len - 1;
        score_f = 0;
        score_r = 0;
        forward_max_ak = 0;
        reverse_max_ak = 0;

        breakpoint_score = INT_MAX;

        finish = false;
        alignment_k = text_len - pattern_len;

        int iteration_count = 0;
        const int max_iterations = max_alignment_steps; 

        extend_max(&finish, score_f, &forward_max_ak, &wf_f, max_score_scope, alignment_k, (int32_t)text_len, pattern_len);
        if(finish) {
            *(breakpoint_score_g + blockIdx.x) = breakpoint_score;
            return;
        }
        
        extend_max(&finish, score_r, &reverse_max_ak, &wf_r, max_score_scope, alignment_k, (int32_t)text_len, pattern_len);
        if(finish) {
            *(breakpoint_score_g + blockIdx.x) = breakpoint_score;
            return;
        }

        int max_ak;
        bool last_wf_forward;
        max_ak = 0;
        last_wf_forward = false;
        
        while (true) {
            iteration_count++;
            if (iteration_count > max_iterations) break;
            
            if (forward_max_ak + reverse_max_ak >= max_antidiag) break;
            
            nextWF(&score_f, &wf_f, true, max_score_scope, text_len, pattern_len, matrix_wf_m_f + matrix_block_offset, 
                  matrix_wf_i_f + matrix_block_offset, matrix_wf_d_f + matrix_block_offset);
            
            extend_max(&finish, score_f, &max_ak, &wf_f, max_score_scope, alignment_k, (int32_t) text_len, pattern_len);
            if (forward_max_ak < max_ak) forward_max_ak = max_ak;
            last_wf_forward = true;
            
            if (forward_max_ak + reverse_max_ak >= max_antidiag) break;
            
            nextWF(&score_r, &wf_r, false, max_score_scope, text_len, pattern_len, matrix_wf_m_r + matrix_block_offset, 
                  matrix_wf_i_r + matrix_block_offset, matrix_wf_d_r + matrix_block_offset);
            
            extend_max(&finish, score_r, &max_ak, &wf_r, max_score_scope, alignment_k, (int32_t) text_len, pattern_len);
            if (reverse_max_ak < max_ak) reverse_max_ak = max_ak;
            last_wf_forward = false;
        }

        int min_score_f, min_score_r;
        while (true) {
            iteration_count++;
            if (iteration_count > max_iterations) break;
            
            if (last_wf_forward) {
                min_score_r = (score_r > max_score_scope - 1) ? score_r - (max_score_scope - 1) : 0;
                if (score_f + min_score_r - penalty_gap_open >= breakpoint_score) break;
                
                overlap(score_f, &wf_f, score_r, &wf_r, max_score_scope, &breakpoint_score, text_len, pattern_len);
                
                nextWF(&score_r, &wf_r, true, max_score_scope, text_len, pattern_len, matrix_wf_m_r + matrix_block_offset, 
                      matrix_wf_i_r + matrix_block_offset, matrix_wf_d_r + matrix_block_offset);
                
                extend(&finish, score_r, &wf_r, max_score_scope, alignment_k, (int32_t) text_len, pattern_len);
            }

            min_score_f = (score_f > max_score_scope - 1) ? score_f - (max_score_scope - 1) : 0;
            if (min_score_f + score_r - penalty_gap_open >= breakpoint_score) break;
            
            overlap(score_r, &wf_r, score_f, &wf_f, max_score_scope, &breakpoint_score, text_len, pattern_len);
            
            nextWF(&score_f, &wf_f, false, max_score_scope, text_len, pattern_len, matrix_wf_m_f + matrix_block_offset, 
                  matrix_wf_i_f + matrix_block_offset, matrix_wf_d_f + matrix_block_offset);
            
            extend(&finish, score_f, &wf_f, max_score_scope, alignment_k, (int32_t) text_len, pattern_len);

            if (score_r + score_f >= max_alignment_steps) break;
            last_wf_forward = true;
        }

        breakpoint_score = -breakpoint_score;
        *(breakpoint_score_g + blockIdx.x) = breakpoint_score;
    }
}

int main(int argc, char *argv[]) {
    if(argc != 2) {
        printf("Error\n");
        return 1;
    }

    FILE *fp = fopen(argv[1], "r");
    if(fp == NULL) {
        printf("File open error\n");
        return 1;
    }

    int num_alignments;
    fscanf(fp, "%d", &num_alignments);
    
    int *pattern_len_array = (int *)malloc(sizeof(int) * num_alignments);
    int *text_len_array = (int *)malloc(sizeof(int) * num_alignments);
    
    int *pattern_offsets = (int *)malloc(sizeof(int) * num_alignments);
    int *text_offsets = (int *)malloc(sizeof(int) * num_alignments);
    
    int *pattern_len_array_g, *text_len_array_g;
    int *pattern_offsets_g, *text_offsets_g;
    
    int total_pattern_len = 0;
    int total_text_len = 0;
    
    for (int i = 0; i < num_alignments; i++) {
        fscanf(fp, "%d", &pattern_len_array[i]);
        fscanf(fp, "%d", &text_len_array[i]);
        
        pattern_offsets[i] = total_pattern_len;
        text_offsets[i] = total_text_len;
        
        total_pattern_len += pattern_len_array[i];
        total_text_len += text_len_array[i];
    }
    
    int *breakpoint_score, *breakpoint_score_g;
    char *pattern_f, *text_f, *pattern_r, *text_r;
    char *pattern_f_g, *text_f_g, *pattern_r_g, *text_r_g;
    wf_t *mwavefronts_f, *iwavefronts_f, *dwavefronts_f;
    wf_t *mwavefronts_r, *iwavefronts_r, *dwavefronts_r;
    int32_t *matrix_wf_m_f, *matrix_wf_i_f, *matrix_wf_d_f, *matrix_wf_m_r, *matrix_wf_i_r, *matrix_wf_d_r;

    pattern_f = (char *)malloc(sizeof(char) * total_pattern_len);
    text_f = (char *)malloc(sizeof(char) * total_text_len);
    
    for (int i = 0; i < num_alignments; i++) {
        fscanf(fp, "%s", pattern_f + pattern_offsets[i]);
        fscanf(fp, "%s", text_f + text_offsets[i]);
    }

    pattern_r = (char *)malloc(sizeof(char) * total_pattern_len);
    text_r = (char *)malloc(sizeof(char) * total_text_len);
    
    for (int j = 0; j < num_alignments; j++) {
        int pattern_len = pattern_len_array[j];
        int text_len = text_len_array[j];
        int pattern_offset = pattern_offsets[j];
        int text_offset = text_offsets[j];
        
        for (int i = 0; i < pattern_len; i++) {
            pattern_r[pattern_offset + i] = pattern_f[pattern_offset + pattern_len - 1 - i];
        }
        
        for (int i = 0; i < text_len; i++) {
            text_r[text_offset + i] = text_f[text_offset + text_len - 1 - i];
        }
    }

    breakpoint_score = (int *)malloc(sizeof(int) * num_alignments);
    for (int i = 0; i < num_alignments; i++) {
        breakpoint_score[i] = INT_MAX;
    }

    int max_score_scope_indel = MAX(penalty_gap_open + penalty_gap_ext, penalty_mismatch) + 1;
    int max_score_scope = MAX(max_score_scope_indel, penalty_mismatch) + 1;

    int hi = 0;
    int lo = 0;
    int eff_lo = lo - (max_score_scope + 1);
    int eff_hi = hi + (max_score_scope + 1);
    lo = MIN(eff_lo, 0);
    hi = MAX(eff_hi, 0);

    int32_t *offsets, *offsets_g;
    offsets = (int32_t *)malloc(sizeof(int32_t) * wf_length);
    for(int i = 0; i < wf_length; i++) {
        offsets[i] = OFFSET_NULL;
    }

    CHECK(hipSetDevice(0));
    
    CHECK(hipMalloc(&pattern_f_g, sizeof(char) * total_pattern_len));
    CHECK(hipMalloc(&pattern_r_g, sizeof(char) * total_pattern_len));
    CHECK(hipMalloc(&text_f_g, sizeof(char) * total_text_len));
    CHECK(hipMalloc(&text_r_g, sizeof(char) * total_text_len));
    
    CHECK(hipMalloc(&pattern_len_array_g, sizeof(int) * num_alignments));
    CHECK(hipMalloc(&text_len_array_g, sizeof(int) * num_alignments));
    CHECK(hipMalloc(&pattern_offsets_g, sizeof(int) * num_alignments));
    CHECK(hipMalloc(&text_offsets_g, sizeof(int) * num_alignments));
    
    CHECK(hipMalloc(&breakpoint_score_g, sizeof(int) * num_alignments));
    CHECK(hipMalloc(&matrix_wf_m_f, sizeof(int32_t) * num_wavefronts * wf_length * num_alignments));
    CHECK(hipMalloc(&matrix_wf_i_f, sizeof(int32_t) * num_wavefronts * wf_length * num_alignments));
    CHECK(hipMalloc(&matrix_wf_d_f, sizeof(int32_t) * num_wavefronts * wf_length * num_alignments));
    CHECK(hipMalloc(&matrix_wf_m_r, sizeof(int32_t) * num_wavefronts * wf_length * num_alignments));
    CHECK(hipMalloc(&matrix_wf_i_r, sizeof(int32_t) * num_wavefronts * wf_length * num_alignments));
    CHECK(hipMalloc(&matrix_wf_d_r, sizeof(int32_t) * num_wavefronts * wf_length * num_alignments));
    CHECK(hipMalloc(&mwavefronts_f, sizeof(wf_t) * num_wavefronts * num_alignments));
    CHECK(hipMalloc(&iwavefronts_f, sizeof(wf_t) * num_wavefronts * num_alignments));
    CHECK(hipMalloc(&dwavefronts_f, sizeof(wf_t) * num_wavefronts * num_alignments));
    CHECK(hipMalloc(&mwavefronts_r, sizeof(wf_t) * num_wavefronts * num_alignments));
    CHECK(hipMalloc(&iwavefronts_r, sizeof(wf_t) * num_wavefronts * num_alignments));
    CHECK(hipMalloc(&dwavefronts_r, sizeof(wf_t) * num_wavefronts * num_alignments));
    CHECK(hipMalloc(&offsets_g, sizeof(int32_t) * wf_length));

    CHECK(hipMemcpy(breakpoint_score_g, breakpoint_score, sizeof(int) * num_alignments, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(pattern_f_g, pattern_f, sizeof(char) * total_pattern_len, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(pattern_r_g, pattern_r, sizeof(char) * total_pattern_len, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(text_f_g, text_f, sizeof(char) * total_text_len, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(text_r_g, text_r, sizeof(char) * total_text_len, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(pattern_len_array_g, pattern_len_array, sizeof(int) * num_alignments, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(text_len_array_g, text_len_array, sizeof(int) * num_alignments, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(pattern_offsets_g, pattern_offsets, sizeof(int) * num_alignments, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(text_offsets_g, text_offsets, sizeof(int) * num_alignments, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(offsets_g, offsets, sizeof(int32_t) * wf_length, hipMemcpyHostToDevice));

    dim3 blocksPerGrid(num_alignments, 1, 1);
    dim3 threadsPerBlock(NUM_THREADS, 1, 1);

    std::chrono::high_resolution_clock::time_point start = NOW;
    
    biWFA_kernel<<<blocksPerGrid, threadsPerBlock>>>(
        pattern_f_g, text_f_g, pattern_r_g, text_r_g, 
        breakpoint_score_g, 
        mwavefronts_f, iwavefronts_f, dwavefronts_f, 
        mwavefronts_r, iwavefronts_r, dwavefronts_r, 
        lo, hi, offsets_g, 
        pattern_len_array_g, text_len_array_g,  
        pattern_offsets_g, text_offsets_g,      
        max_score_scope, 
        matrix_wf_m_f, matrix_wf_i_f, matrix_wf_d_f, 
        matrix_wf_m_r, matrix_wf_i_r, matrix_wf_d_r
    );

    CHECK_KERNELCALL();
    CHECK(hipGetLastError());
    CHECK(hipDeviceSynchronize());

    std::chrono::high_resolution_clock::time_point end = NOW;
    std::chrono::duration<double> time_temp = (end - start);

    CHECK(hipMemcpy(breakpoint_score, breakpoint_score_g, sizeof(int) * num_alignments, hipMemcpyDeviceToHost));
    
    long double total_cells = 0;
    for (int i = 0; i < num_alignments; i++) {
        total_cells += pattern_len_array[i] * text_len_array[i];
    }
    
    long double gcups = total_cells;
    gcups /= 1E9;
    gcups /= time_temp.count();
    
    printf("GPU Time: %lf\n", time_temp.count());
    printf("Estimated GCUPS GPU: : %Lf\n", gcups);
    printf("\n");

    CHECK(hipFree(pattern_f_g));
    CHECK(hipFree(pattern_r_g));
    CHECK(hipFree(text_f_g));
    CHECK(hipFree(text_r_g));
    CHECK(hipFree(pattern_len_array_g));
    CHECK(hipFree(text_len_array_g));
    CHECK(hipFree(pattern_offsets_g));
    CHECK(hipFree(text_offsets_g));
    CHECK(hipFree(breakpoint_score_g));
    CHECK(hipFree(matrix_wf_m_f));
    CHECK(hipFree(matrix_wf_i_f));
    CHECK(hipFree(matrix_wf_d_f));
    CHECK(hipFree(matrix_wf_m_r));
    CHECK(hipFree(matrix_wf_i_r));
    CHECK(hipFree(matrix_wf_d_r));
    CHECK(hipFree(mwavefronts_f));
    CHECK(hipFree(iwavefronts_f));
    CHECK(hipFree(dwavefronts_f));
    CHECK(hipFree(mwavefronts_r));
    CHECK(hipFree(iwavefronts_r));
    CHECK(hipFree(dwavefronts_r));
    CHECK(hipFree(offsets_g));

    printf("Alignment scores:\n\n");
    for (int i = 0; i < num_alignments; i++) {
        int pattern_len = pattern_len_array[i];
        int text_len = text_len_array[i];
        int pattern_offset = pattern_offsets[i];
        int text_offset = text_offsets[i];
        
        printf("%.*s\n%.*s : %d\n", 
               pattern_len, &pattern_f[pattern_offset], 
               text_len, &text_f[text_offset], 
               -breakpoint_score[i]);
        printf("\n");
    }

    printf("Checking alignment scores\n");
    
    wavefront_aligner_attr_t attributes = wavefront_aligner_attr_default;
    attributes.distance_metric = gap_affine;
    attributes.affine_penalties.mismatch = 4;
    attributes.affine_penalties.gap_opening = 6;
    attributes.affine_penalties.gap_extension = 2;
    
    wavefront_aligner_t* wf_aligner = wavefront_aligner_new(&attributes);
    
    bool all_correct = true;
    
    for (int i = 0; i < num_alignments; i++) {
        int pattern_len = pattern_len_array[i];
        int text_len = text_len_array[i];
        int pattern_offset = pattern_offsets[i];
        int text_offset = text_offsets[i];
        
        const char* pattern = &pattern_f[pattern_offset];
        const char* text = &text_f[text_offset];
    
        wavefront_align(wf_aligner, pattern, pattern_len, text, text_len);
        int cpu_score = wf_aligner->cigar->score;
        int gpu_score = breakpoint_score[i];  
    
        printf("Alignment %d - CPU score: %d | GPU score: %d\n", i, cpu_score, gpu_score);
    
        if (cpu_score != gpu_score) {
            printf("ERROR on alignment %d: CPU %d != GPU %d\n", i, cpu_score, gpu_score);
            all_correct = false;
        }
    }

    free(pattern_f);
    free(text_f);
    free(pattern_r);
    free(text_r);
    free(breakpoint_score);
    free(offsets);
    free(pattern_len_array);
    free(text_len_array);
    free(pattern_offsets);
    free(text_offsets);

    wavefront_aligner_delete(wf_aligner); 
    
    fclose(fp);
    return 0;
}
